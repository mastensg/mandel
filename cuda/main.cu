
#include <hip/hip_runtime.h>
#include <cstdio>

#include <hip/hip_runtime_api.h>

__global__ void f(int *x) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (0 == i % 2) {
    x[i] = i;
  }
  // printf("%d\n",i);
}

int main() {
  // const size_t block_size = 1024;
  // const size_t blocks = 1024;
  const size_t block_size = 4;
  const size_t blocks = 4;
  const size_t n = blocks * block_size;
  const size_t size = n * sizeof(int);

  int *h_x = reinterpret_cast<int *>(malloc(size));

  int *d_x;
  hipMalloc(&d_x, size);

  f<<<blocks, block_size>>>(d_x);
  // cudaDeviceSynchronize();

  hipMemcpy(h_x, d_x, size, hipMemcpyDeviceToHost);

#if 1
  for (size_t i = 0; i < n; ++i) {
    fprintf(stderr, "%3zu %3d\n", i, h_x[i]);
  }
#endif
  // std::fprintf(stderr, "%d\n", h_x[n - 1]);

  hipProfilerStop();
}
